#include "hip/hip_runtime.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

// Simple 3D volume renderer

#include <iostream>
#include <stdio.h>

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#include "Constants.h"
#include "Renderable.h"
#include "SdogDB.h"

#include <glm/gtx/intersect.hpp>

typedef unsigned int  uint;
typedef unsigned char uchar;

typedef unsigned char VolumeType;

typedef struct
{
	float4 m[3];
} float3x4;

typedef struct
{
	float4 m[4];
} float4x4;

enum SdogCellType {
	NG,
	LG,
	SG,
	INVALID
};

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix


__device__ struct SphereCoord
{
	SphereCoord(float x, float y, float z) {
		radius = glm::length(glm::vec3(x, y, z));
		latitude = asin(y / radius);
		longitude = atan2(x, z);
	};

	double latitude;
	double longitude;
	double radius;
};

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
	float3 r;
	r.x = dot(v, make_float3(M.m[0]));
	r.y = dot(v, make_float3(M.m[1]));
	r.z = dot(v, make_float3(M.m[2]));
	return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
	float4 r;
	r.x = dot(v, M.m[0]);
	r.y = dot(v, M.m[1]);
	r.z = dot(v, M.m[2]);
	r.w = 1.0f;
	return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float4x4 &M, const float4 &v)
{
	float4 r;
	r.x = dot(v, M.m[0]);
	r.y = dot(v, M.m[1]);
	r.z = dot(v, M.m[2]);
	r.w = dot(v, M.m[3]);
	return r;
}


__device__ uint rgbaFloatToInt(float4 rgba)
{
	rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
	rgba.y = __saturatef(rgba.y);
	rgba.z = __saturatef(rgba.z);
	rgba.w = __saturatef(rgba.w);
	return (uint(rgba.w * 255) << 24) | (uint(rgba.z * 255) << 16) | (uint(rgba.y * 255) << 8) | uint(rgba.x * 255);
}

//__device__ std::string code = SdogCell::codeForPos(coord.latitude, coord.longitude, coord.radius, (RADIUS_EARTH_KM * 4.f / 3.f), 8);

__global__ void
d_render(uint *d_output, uint imageW, uint imageH, float4x4 invProjView, float4x4 invWorldModel, float3 camPos, SdogDB* database)
{

	//uint x = blockIdx.x*blockDim.x + threadIdx.x;
	//uint y = blockIdx.y*blockDim.y + threadIdx.y;

	//if ((x >= imageW) || (y >= imageH)) return;

	//float4 worldNew = make_float4(x, y, -1.f, 1.f);

	//worldNew = mul(invProjView, worldNew);

	//worldNew.x /= worldNew.w;
	//worldNew.y /= worldNew.w;
	//worldNew.z /= worldNew.w;

	//worldNew = mul(invWorldModel, worldNew);

	//float3 rayO = camPos;

	//float3 rayD = normalize(make_float3(worldNew.x, worldNew.y, worldNew.z) - rayO);

	//float4 colour = make_float4(0.f, 0.f, 0.f, 0.f);

	//int continueCount = 0;
	//int count = 0;
	//int k = 0;
	//std::string prevCode = "";

	//while (count < 750)
	//{
	//	float3 tracePoint = rayO + ((float)k * normalize(rayD));

	//	k += 50;

	//	//glm::vec4 t = inverseModel * glm::vec4(tracePoint, 1.0);
	//	//tracePoint = glm::vec3(t.x, t.y, t.z);

	//	SphereCoord coord(tracePoint.x, tracePoint.y, tracePoint.z);

	//	std::string code = SdogCell::codeForPos(coord.latitude, coord.longitude, coord.radius, (RADIUS_EARTH_KM * 4.f / 3.f), 8);

	//	std::vector<AirSigmet> sigs;

	//	if (code.compare(prevCode) == 0) {
	//		continueCount++;
	//		if (continueCount > 1000)
	//		{
	//			//std::cout << "too many continues" << std::endl;

	//			break;
	//		}
	//		//std::cout << "continuing" << std::endl;
	//		continue;
	//	}
	//	else if (strtol(code.c_str(), (char**)NULL, 10) == 0) break;

	//	continueCount = 0;
	//	//std::cout << code << std::endl; 

	//	prevCode = code;
	//	count++;

	//	database->getAirSigmetForCell(code, sigs);

	//	if (sigs.size() == 0) continue;

	//	//std::cout << "has sigmet" << std::endl;

	//	colour = make_float4(1.f, 0.f, 0.f, 0.5f);
	//	break;

	//}

	//// write output color
	//d_output[y*imageW + x] = rgbaFloatToInt(colour);
}

extern "C"
void initCuda(void *h_volume, hipExtent volumeSize)
{
	// create 3D array
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
	//checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

	// copy data to 3D array
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr(h_volume, volumeSize.width * sizeof(VolumeType), volumeSize.width, volumeSize.height);
	//copyParams.dstArray = d_volumeArray;
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParams));

	// bind array to 3D texture
	//checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));

	// create transfer function texture
	float4 transferFunc[] =
	{
		{ 0.0, 0.0, 0.0, 0.0, },
	{ 1.0, 0.0, 0.0, 1.0, },
	{ 1.0, 0.5, 0.0, 1.0, },
	{ 1.0, 1.0, 0.0, 1.0, },
	{ 0.0, 1.0, 0.0, 1.0, },
	{ 0.0, 1.0, 1.0, 1.0, },
	{ 0.0, 0.0, 1.0, 1.0, },
	{ 1.0, 0.0, 1.0, 1.0, },
	{ 0.0, 0.0, 0.0, 0.0, },
	};

	hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
	hipArray *d_transferFuncArray;
	checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc) / sizeof(float4), 1));
	checkCudaErrors(hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice));
}


extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, uint imageW, uint imageH, float4x4 invProjView,
	float4x4 invWorldModel, float3 camPos, SdogDB* database)
{
	d_render << <gridSize, blockSize >> > (d_output, imageW, imageH, invProjView, invWorldModel, camPos, database);
}

// These two functions from https://stackoverflow.com/questions/20201335/add-char-arrays-in-cuda
__device__ char * my_strcpy(char dest[], const char src[]) {
	int i = 0;
	do {
		dest[i] = src[i];
	} while (src[i++] != 0);
	return dest;
}

__device__ char* my_strcat(char dest[], const char src[]) {
	int i = 0;
	while (dest[i] != 0) i++;
	my_strcpy(dest + i, src);
	return dest;
}

// Returns the SDOG cell code for the provided spherical point at a given subdivision level - Converted to CUDA code from Ben's C++ function
__global__ void
cudaCodeForPos_kernel(double latRad, double longRad, double radius, double gridRadius, unsigned int level, char* returnCode) {

	double M_PI = 3.14159265358979323846;
	double M_PI_2 = M_PI / 2.0;

	char code[15];
	if (longRad < -M_PI) {
		longRad += 2.0 * M_PI;
	}
	if (longRad > M_PI) {
		longRad -= 2.0 * M_PI;
	}

	double minLat, maxLat, minLong, maxLong, minRad, maxRad;
	minLat = 0.0;
	maxLat = M_PI_2;
	minRad = 0.0;
	maxRad = gridRadius;

	// Determine which otcant the point is in
	char octCode = '0';
	if (latRad < 0.0) {
		octCode = '4';
		//octCode = 4;
	}
	if (longRad < 0.0) {
		if (octCode == '0') {
			octCode = '2';
		}
		else if (octCode == '4') {
			octCode = '6';
		}
		//octCode += 2;
	}
	if (abs(longRad) > M_PI_2) {
		if (octCode == '0') {
			octCode = '1';
		}
		else if (octCode == '4') {
			octCode = '5';
		}
		else if (octCode == '2') {
			octCode = '3';
		}
		else if (octCode == '6') {
			octCode = '7';
		}

		//octCode += 1;
		minLong = M_PI_2;
		maxLong = M_PI;
	}
	else {
		minLong = 0.0;
		maxLong = M_PI_2;
	}
	latRad = abs(latRad);
	longRad = abs(longRad);

	code[0] = octCode;

	// Loop for desired number of levels and determine
	// which child point is in for each itteration
	SdogCellType curType = SdogCellType::SG;
	for (unsigned int i = 0; i < level; i++) {

		char childCode = '0';
		double midLat = 0.5 * minLat + 0.5 * maxLat;
		double midLong = 0.5 * minLong + 0.5 * maxLong;
		double midRad = 0.5 * minRad + 0.5 * maxRad;

		if (curType == SdogCellType::NG) {

			if (radius > midRad) {
				minRad = midRad;
			}
			else {
				childCode = '4';
				//childCode += 4;
				maxRad = midRad;
			}
			if (latRad < midLat) {
				maxLat = midLat;
			}
			else {
				if (childCode == '0') {
					childCode = '2';
				}
				else childCode = '6';
				//childCode += 2;
				minLat = midLat;
			}
			if (longRad < midLong) {
				maxLong = midLong;
			}
			else {
				if (childCode == '0') {
					childCode = '1';
				}
				else if (childCode == '2') {
					childCode = '3';
				}
				else if (childCode == '4') {
					childCode = '5';
				}
				else if (childCode == '6') {
					childCode = '7';
				}
				//childCode += 1;
				minLong = midLong;
			}
			// type doesn't change
		}
		// childCode = '0'
		else if (curType == SdogCellType::LG) {

			if (radius > midRad) {
				minRad = midRad;
			}
			else {
				maxRad = midRad;
				childCode = '3';
				//childCode += 3;
			}
			if (latRad < midLat) {
				maxLat = midLat;
				curType = SdogCellType::NG;

				if (longRad < midLong) {
					maxLong = midLong;
				}
				else {
					if (childCode == '0') {
						childCode = '1';
					}
					else if (childCode == '3') {
						childCode = '4';
					}
					//childCode += 1;
					minLong = midLong;
				}
			}
			// childCode = 0 or 3
			else {
				if (childCode == '0') {
					childCode = '2';
				}
				else if (childCode == '3') {
					childCode = '5';
				}
				//childCode += 2;
				minLat = midLat;
				// type doesn't change
			}
		}
		// childCode = '0'
		else {// curType == SdogCellType::SG

			if (radius > midRad) {

				minRad = midRad;

				if (latRad < midLat) {
					maxLat = midLat;
					curType = SdogCellType::NG;

					if (longRad < midLong) {
						childCode = '0';
						maxLong = midLong;
					}
					else {
						childCode = '1';
						minLong = midLong;
					}
				}
				else {
					childCode = '2';
					minLat = midLat;
					curType = SdogCellType::LG;
				}
			}
			else {
				childCode = '3';
				maxRad = midRad;
				// type doesn't change
			}
		}
		code[i + 1] = childCode;
	}

	int i = 0;
	do {
		returnCode[i] = code[i];
	} while (code[i++] != 0);
}

// latRad - latitude of point, in radians
// longRad - longitude of point, in radians
// radius - radius of point (0 is centre of the sphere)
// gridRadius - radius of the grid the cell belongs to
// return - code for the cell that contains the point
extern "C"
void cudaCodeForPos(double latRad, double longRad, double radius, double gridRadius, unsigned int level, char* returnCode) {
	cudaCodeForPos_kernel << <1, 1 >> >(latRad, longRad, radius, gridRadius, level, returnCode);
}

__global__ void
getInteger_kernel(uint* returnInt) {
	returnInt[0] = 12;
}

extern "C"
void getInteger(uint* returnInt) {
	getInteger_kernel <<<1,1>>>(returnInt);
}

#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
